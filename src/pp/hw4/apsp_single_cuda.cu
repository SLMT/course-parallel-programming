#include "hip/hip_runtime.h"
#include "apsp.hpp"

namespace pp {
namespace hw4 {

__device__ void CalcABlock(Cost *self, Cost *depen1, Cost *depen2, unsigned block_size, unsigned num_mid) {
	// Plan: We can map 1 APSP block to 1 CUDA block.
	// A value of a block is assigned to a CUDA thread of a CUDA block.
	// It needs to be looped k times for k middle nodes.
	// Each loop should have a synchronized barrier in the end.

	Cost cost1, cost2, final_cost;
	unsigned src = threadIdx.x;
	unsigned dst = threadIdx.y;

	for (unsigned mid = 0; mid < num_mid; mid++) {
		// Find the smaller cost
		cost1 = self[src * block_size + dst];
		cost2 = depen1[src * block_size + mid] + depen2[mid * block_size + dst];
		final_cost = (cost1 < cost2)? cost1 : cost2;

		// Synchronized
		__syncthreads();

		// Save the new cost back to shared memory
		self[src * block_size + dst] = final_cost;

		// Synchronized
		__syncthreads();
	}
}

// A shared memory variable
extern __shared__ Cost costs_in_sm[];

__device__ void CopyCostFromGlobalToSM(Cost *gl, Cost *sm, unsigned num_nodes, unsigned block_size, unsigned bx, unsigned by) {
	unsigned gx = bx * block_size + threadIdx.x;
	unsigned gy = by * block_size + threadIdx.y;
	unsigned sx = threadIdx.x;
	unsigned sy = threadIdx.y;

	if (gx < num_nodes && gy < num_nodes)
		sm[sx * block_size + sy] = gl[gx * num_nodes + gy];
}

__device__ void CopyCostFromSMToGlobal(Cost *gl, Cost *sm, unsigned num_nodes, unsigned block_size, unsigned bx, unsigned by) {
	unsigned gx = bx * block_size + threadIdx.x;
	unsigned gy = by * block_size + threadIdx.y;
	unsigned sx = threadIdx.x;
	unsigned sy = threadIdx.y;

	if (gx < num_nodes && gy < num_nodes)
		gl[gx * num_nodes + gy] = sm[sx * block_size + sy];
}

__global__ void CalcIndependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx) {
	unsigned gx = round_idx * block_size + threadIdx.x;
	unsigned gy = round_idx * block_size + threadIdx.y;
	unsigned num_mid = ((round_idx + 1) * block_size < num_nodes)? block_size : num_nodes - round_idx * block_size;

	// Move the data from Global to Shared Memory
	CopyCostFromGlobalToSM(costs, costs_in_sm, num_nodes, block_size, round_idx, round_idx);

	// Calculate the block
	if (gx < num_nodes && gy < num_nodes)
		CalcABlock(costs_in_sm, costs_in_sm, costs_in_sm, block_size, num_mid);

	// Move the data back to Global
	CopyCostFromSMToGlobal(costs, costs_in_sm, num_nodes, block_size, round_idx, round_idx);
}

__global__ void CalcSinglyDependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx, unsigned block_x_start, unsigned block_y_start) {
	unsigned gx = (block_x_start + blockIdx.x) * block_size + threadIdx.x;
	unsigned gy = (block_y_start + blockIdx.y) * block_size + threadIdx.y;
	unsigned num_mid = ((round_idx + 1) * block_size < num_nodes)? block_size : num_nodes - round_idx * block_size;
	Cost *self_in_sm = costs_in_sm;
	Cost *depen_in_sm = costs_in_sm + block_size * block_size;

	// Move the data from Global to Shared Memory
	CopyCostFromGlobalToSM(costs, self_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, block_y_start + blockIdx.y);
	CopyCostFromGlobalToSM(costs, depen_in_sm, num_nodes, block_size, round_idx, round_idx);

	// Calculate the block
	if (gx < num_nodes && gy < num_nodes)
		CalcABlock(self_in_sm, depen_in_sm, self_in_sm, block_size, num_mid);

	// Move the data back to Global
	CopyCostFromSMToGlobal(costs, self_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, block_y_start + blockIdx.y);
	CopyCostFromSMToGlobal(costs, depen_in_sm, num_nodes, block_size, round_idx, round_idx);
}

__global__ void CalcDoublyDependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx, unsigned block_x_start, unsigned block_y_start) {
	unsigned gx = (block_x_start + blockIdx.x) * block_size + threadIdx.x;
	unsigned gy = (block_y_start + blockIdx.y) * block_size + threadIdx.y;
	unsigned num_mid = ((round_idx + 1) * block_size < num_nodes)? block_size : num_nodes - round_idx * block_size;
	Cost *self_in_sm = costs_in_sm;
	Cost *depen1_in_sm = costs_in_sm + block_size * block_size;
	Cost *depen2_in_sm = costs_in_sm + 2 * block_size * block_size;

	// Move the data from Global to Shared Memory
	CopyCostFromGlobalToSM(costs, self_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, block_y_start + blockIdx.y);
	CopyCostFromGlobalToSM(costs, depen1_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, round_idx);
	CopyCostFromGlobalToSM(costs, depen2_in_sm, num_nodes, block_size, round_idx, block_y_start + blockIdx.y);

	// Calculate the block
	if (gx < num_nodes && gy < num_nodes)
		CalcABlock(self_in_sm, depen1_in_sm, depen2_in_sm, block_size, num_mid);

	// Move the data back to Global
	CopyCostFromSMToGlobal(costs, self_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, block_y_start + blockIdx.y);
	CopyCostFromSMToGlobal(costs, depen1_in_sm, num_nodes, block_size, block_x_start + blockIdx.x, round_idx);
	CopyCostFromSMToGlobal(costs, depen2_in_sm, num_nodes, block_size, round_idx, block_y_start + blockIdx.y);
}

void CUDACalcIndependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx) {
	dim3 num_blocks(1, 1);
	dim3 num_threads(block_size, block_size);
	unsigned sm_size = block_size * block_size * sizeof(Cost);

	CalcIndependBlocks<<<num_blocks, num_threads, sm_size>>>(costs, num_nodes, block_size, round_idx);
}

void CUDACalcSinglyDependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx, unsigned block_x_start, unsigned block_y_start, unsigned block_x_len, unsigned block_y_len) {
	dim3 num_blocks(block_x_len, block_y_len);
	dim3 num_threads(block_size, block_size);
	unsigned sm_size = 2 * block_size * block_size * sizeof(Cost);

	CalcSinglyDependBlocks<<<num_blocks, num_threads, sm_size>>>(costs, num_nodes, block_size, round_idx, block_x_start, block_y_start);
}

void CUDACalcDoublyDependBlocks(Cost *costs, unsigned num_nodes, unsigned block_size, unsigned round_idx, unsigned block_x_start, unsigned block_y_start, unsigned block_x_len, unsigned block_y_len) {
	dim3 num_blocks(block_x_len, block_y_len);
	dim3 num_threads(block_size, block_size);
	unsigned sm_size = 3 * block_size * block_size * sizeof(Cost);

	CalcDoublyDependBlocks<<<num_blocks, num_threads, sm_size>>>(costs, num_nodes, block_size, round_idx, block_x_start, block_y_start);
}


void CalcAPSP(Graph *graph, unsigned block_size) {
	unsigned nvertices = graph->num_vertices;

	// Device (GPU) Initialization
	//hipSetDevice(0);

	// Allocate memory on GPU
	Cost *costs_on_gpu;
	unsigned data_size = sizeof(Cost) * nvertices * nvertices;
	hipMalloc((void **) &costs_on_gpu, data_size);

	// XXX: Debug
	// printf("Original:\n");
	// PrintCosts(stdout, graph);
	// printf("\n");

	// Copy the graph from Host to Device
	hipMemcpy(costs_on_gpu, graph->weights, data_size, hipMemcpyHostToDevice);

	// Blocked-APSP Algorithm
	unsigned num_rounds = (nvertices % block_size == 0)? nvertices / block_size : nvertices / block_size + 1;
	for (unsigned round_idx = 0; round_idx < num_rounds; round_idx++) {
		unsigned rp1 = round_idx + 1;
		unsigned rr1 = num_rounds - round_idx - 1;

		// Phase 1
		CUDACalcIndependBlocks(costs_on_gpu, nvertices, block_size, round_idx);
		// Wait for complete
		hipDeviceSynchronize();

		// XXX: Debug
		// hipMemcpy(graph->weights, costs_on_gpu, data_size, hipMemcpyDeviceToHost);
		// printf("Round %u, phase 1:\n", round_idx);
		// PrintCosts(stdout, graph);
		// printf("\n");

		// Phase 2
		// Up
		CUDACalcSinglyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, round_idx, 0, 1, round_idx);
		// Left
		CUDACalcSinglyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, round_idx, round_idx, 1);
		// Right
		CUDACalcSinglyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, round_idx, rr1, 1);
		// Down
		CUDACalcSinglyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, round_idx, rp1, 1, rr1);
		// Wait for complete
		hipDeviceSynchronize();

		// XXX: Debug
		// hipMemcpy(graph->weights, costs_on_gpu, data_size, hipMemcpyDeviceToHost);
		// printf("Round %u, phase 2:\n", round_idx);
		// PrintCosts(stdout, graph);
		// printf("\n");

		// Phase 3
		// Left-Up
		CUDACalcDoublyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, 0, round_idx, round_idx);
		// Right-Up
		CUDACalcDoublyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, 0, rr1, round_idx);
		// Left-Down
		CUDACalcDoublyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, rp1, round_idx, rr1);
		// Right-Down
		CUDACalcDoublyDependBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, rp1, rr1, rr1);
		// Wait for complete
		hipDeviceSynchronize();

		// XXX: Debug
		// hipMemcpy(graph->weights, costs_on_gpu, data_size, hipMemcpyDeviceToHost);
		// printf("Round %u, phase 3:\n", round_idx);
		// PrintCosts(stdout, graph);
		// printf("\n");
	}

	// Copy the result from Device to Host
	hipMemcpy(graph->weights, costs_on_gpu, data_size, hipMemcpyDeviceToHost);

	// Free memory on GPU
	hipFree(costs_on_gpu);
}

}
}
