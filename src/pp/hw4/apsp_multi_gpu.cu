#include "apsp.hpp"

#include "block_calculation.hpp"

namespace pp {
namespace hw4 {

void ParallelCalcBlocks() {
	// TODO: Determine a range for calculation

	// TODO: Copy the data from Host to Device

	// TODO: Calculate the blocks in the range

	// TODO: Copy the data from Device to Host

	// TODO: Send the data in its range to other processes (threads)

	// TODO: Receive the data from other processes (threads)
}

void CalcAPSP(Graph *graph, unsigned block_size) {
	unsigned nvertices = graph->num_vertices;

	// Device (GPU) Initialization
	//hipSetDevice(0);

	// Allocate memory on GPU
	Cost *costs_on_gpu;
	unsigned data_size = sizeof(Cost) * nvertices * nvertices;
	hipMalloc((void **) &costs_on_gpu, data_size);

	// Copy the graph from Host to Device
	hipMemcpy(costs_on_gpu, graph->weights, data_size, hipMemcpyHostToDevice);

	// Blocked-APSP Algorithm
	unsigned num_rounds = (nvertices % block_size == 0)? nvertices / block_size : nvertices / block_size + 1;
	for (unsigned round_idx = 0; round_idx < num_rounds; round_idx++) {
		unsigned rp1 = round_idx + 1;
		unsigned rr1 = num_rounds - round_idx - 1;

		// Phase 1
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, round_idx, round_idx, 1, 1);
		// Wait for complete
		hipDeviceSynchronize();

		// Phase 2
		// Up
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, round_idx, 0, 1, round_idx);
		// Left
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, round_idx, round_idx, 1);
		// Right
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, round_idx, rr1, 1);
		// Down
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, round_idx, rp1, 1, rr1);
		// Wait for complete
		hipDeviceSynchronize();

		// Phase 3
		// Left-Up
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, 0, round_idx, round_idx);
		// Right-Up
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, 0, rr1, round_idx);
		// Left-Down
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, 0, rp1, round_idx, rr1);
		// Right-Down
		CalcBlocks(costs_on_gpu, nvertices, block_size, round_idx, rp1, rp1, rr1, rr1);
		// Wait for complete
		hipDeviceSynchronize();
	}

	// Copy the result from Device to Host
	hipMemcpy(graph->weights, costs_on_gpu, data_size, hipMemcpyDeviceToHost);

	// Free memory on GPU
	hipFree(costs_on_gpu);
}

} // namespace hw4
} // namespace pp
